#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <time.h>

// Error checking macro for CUDA calls
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            fprintf(stderr, "CUDA error at %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Queue structure for GPU with fixed-size array
struct Queue {
    int elements[32];  // Fixed size array, assuming k <= 32
    int capacity;
    int size;
    int front;
    int rear;
};

// CUDA kernel for computing k-difference LCP table
__global__ void compute_k_lcp_kernel(
    const char* S1,
    const char* S2,
    int* LCP,
    int n,
    int m,
    int k
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    
    if (i >= n || j >= m) return;

    // Create queue for this thread
    Queue Q;
    Q.capacity = k;
    Q.size = 0;
    Q.front = 0;
    Q.rear = -1;

    int p = 0;
    int max_length = 0;

    // Compare characters and count mismatches
    while ((i + p < n) && (j + p < m)) {
        if (S1[i + p] != S2[j + p]) {
            if (Q.size == k) {
                break;  // Stop when we exceed k mismatches
            }
            // Enqueue
            Q.rear = (Q.rear + 1) % k;
            Q.elements[Q.rear] = p;
            Q.size++;
        }
        p++;
        max_length = p;
    }

    // Store result in flattened array
    LCP[i * m + j] = max_length;
}

// Main function to compute k-difference LCP table using CUDA
int** compute_k_lcp_cuda(const char* S1, const char* S2, int k, double* transfer_time, double* kernel_time) {
    int n = strlen(S1);
    int m = strlen(S2);

    // Allocate device memory
    char* d_S1;
    char* d_S2;
    int* d_LCP;
    int** h_LCP;

    // Start timing memory transfers
    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    CUDA_CHECK(hipEventRecord(start));

    // Use pinned memory for faster transfers
    char* h_S1_pinned, *h_S2_pinned;
    CUDA_CHECK(hipHostMalloc(&h_S1_pinned, n * sizeof(char), hipHostMallocDefault));
    CUDA_CHECK(hipHostMalloc(&h_S2_pinned, m * sizeof(char), hipHostMallocDefault));
    memcpy(h_S1_pinned, S1, n * sizeof(char));
    memcpy(h_S2_pinned, S2, m * sizeof(char));

    CUDA_CHECK(hipMalloc(&d_S1, n * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_S2, m * sizeof(char)));
    CUDA_CHECK(hipMalloc(&d_LCP, n * m * sizeof(int)));

    // Allocate host memory for result
    h_LCP = (int**)malloc(n * sizeof(int*));
    for (int i = 0; i < n; i++) {
        h_LCP[i] = (int*)malloc(m * sizeof(int));
    }

    // Copy input data to device using pinned memory
    CUDA_CHECK(hipMemcpy(d_S1, h_S1_pinned, n * sizeof(char), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_S2, h_S2_pinned, m * sizeof(char), hipMemcpyHostToDevice));

    // Record transfer time
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float transfer_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&transfer_ms, start, stop));
    *transfer_time = transfer_ms / 1000.0;

    // Calculate grid and block dimensions
    dim3 block_size(16, 16);  // Smaller block size for better occupancy
    dim3 grid_size((n + block_size.x - 1) / block_size.x, 
                   (m + block_size.y - 1) / block_size.y);

    // Start timing kernel execution
    CUDA_CHECK(hipEventRecord(start));

    // Launch kernel
    compute_k_lcp_kernel<<<grid_size, block_size>>>(d_S1, d_S2, d_LCP, n, m, k);

    // Record kernel time
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float kernel_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&kernel_ms, start, stop));
    *kernel_time = kernel_ms / 1000.0;

    // Start timing result transfer
    CUDA_CHECK(hipEventRecord(start));

    // Copy result back to host row by row
    for (int i = 0; i < n; i++) {
        CUDA_CHECK(hipMemcpy(h_LCP[i], d_LCP + i * m, m * sizeof(int), hipMemcpyDeviceToHost));
    }

    // Record result transfer time
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float result_transfer_ms = 0;
    CUDA_CHECK(hipEventElapsedTime(&result_transfer_ms, start, stop));
    *transfer_time += result_transfer_ms / 1000.0;

    // Free device memory
    CUDA_CHECK(hipFree(d_S1));
    CUDA_CHECK(hipFree(d_S2));
    CUDA_CHECK(hipFree(d_LCP));
    CUDA_CHECK(hipHostFree(h_S1_pinned));
    CUDA_CHECK(hipHostFree(h_S2_pinned));

    // Clean up CUDA events
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return h_LCP;
}

// CPU implementation of k-difference LCP table
int** compute_k_lcp_cpu(const char* S1, const char* S2, int k) {
    int n = strlen(S1);
    int m = strlen(S2);
    
    // Allocate result array
    int** LCP = (int**)malloc(n * sizeof(int*));
    for (int i = 0; i < n; i++) {
        LCP[i] = (int*)malloc(m * sizeof(int));
    }

    // Compute LCP table
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            Queue Q;
            Q.capacity = k;
            Q.size = 0;
            Q.front = 0;
            Q.rear = -1;

            int p = 0;
            int max_length = 0;

            while ((i + p < n) && (j + p < m)) {
                if (S1[i + p] != S2[j + p]) {
                    if (Q.size == k) {
                        break;
                    }
                    Q.rear = (Q.rear + 1) % k;
                    Q.elements[Q.rear] = p;
                    Q.size++;
                }
                p++;
                max_length = p;
            }

            LCP[i][j] = max_length;
        }
    }

    return LCP;
}

// Function to generate random string
char* generate_random_string(int length) {
    char* str = (char*)malloc((length + 1) * sizeof(char));
    const char charset[] = "abcdefghijklmnopqrstuvwxyz";
    for (int i = 0; i < length; i++) {
        str[i] = charset[rand() % (sizeof(charset) - 1)];
    }
    str[length] = '\0';
    return str;
}

// Function to free 2D array
void free_2d_array(int** arr, int rows) {
    for (int i = 0; i < rows; i++) {
        free(arr[i]);
    }
    free(arr);
}

int main(int argc, char* argv[]) {
    // Initialize random seed
    srand(time(NULL));

    // Generate two random strings of length 5000 (increased for better GPU utilization)
    int str_length = 5000;
    char* str1 = generate_random_string(str_length);
    char* str2 = generate_random_string(str_length);
    int k = 2;

    printf("Benchmarking with strings of length %d\n", str_length);
    printf("k = %d\n", k);

    // CPU Implementation
    clock_t cpu_start = clock();
    int** cpu_result = compute_k_lcp_cpu(str1, str2, k);
    clock_t cpu_end = clock();
    double cpu_time = ((double)(cpu_end - cpu_start)) / CLOCKS_PER_SEC;

    // CUDA Implementation with timing breakdown
    double transfer_time = 0, kernel_time = 0;
    int** cuda_result = compute_k_lcp_cuda(str1, str2, k, &transfer_time, &kernel_time);
    double total_cuda_time = transfer_time + kernel_time;

    // Verify results match
    bool results_match = true;
    for (int i = 0; i < str_length; i++) {
        for (int j = 0; j < str_length; j++) {
            if (cpu_result[i][j] != cuda_result[i][j]) {
                results_match = false;
                break;
            }
        }
        if (!results_match) break;
    }

    // Print results
    printf("\nPerformance Results:\n");
    printf("CPU Time: %.4f seconds\n", cpu_time);
    printf("CUDA Time Breakdown:\n");
    printf("  - Memory Transfer: %.4f seconds\n", transfer_time);
    printf("  - Kernel Execution: %.4f seconds\n", kernel_time);
    printf("  - Total CUDA Time: %.4f seconds\n", total_cuda_time);
    printf("Speedup: %.2fx\n", cpu_time / total_cuda_time);
    printf("Results Match: %s\n", results_match ? "Yes" : "No");

    // Free memory
    free(str1);
    free(str2);
    free_2d_array(cpu_result, str_length);
    free_2d_array(cuda_result, str_length);

    return 0;
} 